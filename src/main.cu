#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include "nfd.h"
#include <iostream>
#include <string>

__global__ void grayscaleConversion(unsigned char *inputImage, unsigned char *outputImage, int width, int height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int index = y * width + x;
        unsigned char r = inputImage[channels * index];
        unsigned char g = inputImage[channels * index + 1];
        unsigned char b = inputImage[channels * index + 2];
        outputImage[index] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

void checkCudaErrors(hipError_t err, const char *message)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << message << " : " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

int main()
{
    NFD_Init();

    nfdchar_t *outPath;
    nfdfilteritem_t filterItem[1] = {{"Image", "png,jpg,jpeg"}};
    nfdresult_t result = NFD_OpenDialog(&outPath, filterItem, 1, NULL);
    if (result == NFD_ERROR)
    {
        std::cerr << "Error: " << NFD_GetError() << std::endl;
        NFD_Quit();
        return -1;
    }

    if (result == NFD_CANCEL)
    {
        NFD_Quit();
        return -1;
    }

    int width, height, channels;
    unsigned char *image_data = stbi_load(outPath, &width, &height, &channels, 0);

    if (!image_data)
    {
        std::cerr << "Failed to load image: " << outPath << std::endl;
        NFD_Quit();
        return -1;
    }

    NFD_FreePath(outPath);

    std::cout << "Loaded image with dimensions: " << width << "x" << height << " and " << channels << " channels." << std::endl;

    unsigned char *gray_output_img = new unsigned char[width * height];

    // Allocate memory on GPU
    unsigned char *d_inputImage, *d_outputImage;
    hipMalloc(&d_inputImage, width * height * 3 * sizeof(unsigned char));
    hipMalloc(&d_outputImage, width * height * sizeof(unsigned char));

    // Transfer input image data to GPU
    hipMemcpy(d_inputImage, image_data, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    stbi_image_free(image_data);

    // Define block and grid dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    grayscaleConversion<<<gridSize, blockSize>>>(d_inputImage, d_outputImage, width, height, channels);
    checkCudaErrors(hipGetLastError(), "Kernel launch failed");
    checkCudaErrors(hipDeviceSynchronize(), "Kernel execution failed");


    // Transfer output image data to CPU
    hipMemcpy(gray_output_img, d_outputImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    result = NFD_SaveDialogU8(&outPath, filterItem, 1, NULL, "gray_scaled");
    if (result == NFD_ERROR)
    {
        std::cerr << "Error: " << NFD_GetError() << std::endl;
        NFD_Quit();
        return -1;
    }

    // Write the image
    if (stbi_write_png(outPath, width, height, 1, gray_output_img, width))
    {
        std::cout << "Grayscale image written successfully: " << outPath << std::endl;
    }
    else
    {
        std::cerr << "Failed to write grayscale image." << std::endl;
    }

    NFD_FreePath(outPath);

    delete[] gray_output_img;

    hipFree(d_inputImage);
    hipFree(d_outputImage);

    NFD_Quit();

    return 0;
}